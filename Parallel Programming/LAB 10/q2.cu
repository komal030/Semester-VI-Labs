#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void calc(int* A,int m,int n)
{
int row_id=threadIdx.x;
for( int i=0;i<n;i++)
{
int ele=A[row_id*n+i];
for(int j=0;j<row_id;j++)
ele=ele*A[row_id*n+i];
A[row_id*n+i]=ele;
}
}


int main()
{
int m,n;
int *A;
printf("Enter number of rows:");
scanf("%d",&m);
printf("Enter number of columns:");
scanf("%d",&n);
int size=m*n*sizeof(int);
A=(int*)malloc(size);
printf("Enter the elements\n");
for(int i=0;i<m*n;i++)
{
scanf("%d",&A[i]);
}

printf("Original Matrix\n");
for(int i=0;i<m*n;i++)
{
if(i%n==0)
printf("\n");
printf("%d ",A[i]);
}
int *cudaA;
hipMalloc(&cudaA,size);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
dim3 dimGrid(1,1,1);
dim3 dimBlock(m,1,1);
calc<<<dimGrid,dimBlock>>>(cudaA,m,n);
hipMemcpy(A,cudaA,size,hipMemcpyDeviceToHost);
printf("\nMofified Matrix\n");
for(int i=0;i<m*n;i++)
{
if(i%n==0)
printf("\n");
printf("%d ",A[i]);
}
}
