#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void calc(int *A,char* B,char *C,int m,int n)
{
int row_id=threadIdx.x;
int col_id=threadIdx.y;
int index=0;
for(int i=0;i<row_id;i++)
{
for(int j=0;j<n;j++)
{
index+=A[i*n+j];
}
}
for(int j=0;j<col_id;j++)
index+=A[row_id*n+j];

for(int i=0;i<A[row_id*n+col_id];i++)
C[index+i]=B[row_id*n+col_id];
}


int main()
{
int m,n;
int *A;
char *B;
char *C;
int *cudaA;
char *cudaB;
char *cudaC;
printf("Enter number of rows:");
scanf("%d",&m);
printf("Enter number of columns:");
scanf("%d",&n);

int size=m*n*sizeof(int);
A=(int*)malloc(m*n*sizeof(int));
B=(char*)malloc(sizeof(char)*m*n);
C=(char*)malloc(sizeof(char)*1024);

printf("Enter the integer elements\n");
for(int i=0;i<m*n;i++)
{
scanf("%d",&A[i]);
}

/*printf("Enter the character elements\n");
for(int i=0;i<m*n;i++)
{
scanf("%c",&B[i]);
}*/

printf("Enter the character elements\n");
scanf("%s",B);


printf("Original Matrix A\n");
for(int i=0;i<m*n;i++)
{
if(i%n==0)
printf("\n");
printf("%d ",A[i]);
}
printf("\nOriginal Matrix B\n");
for(int i=0;i<m*n;i++)
{
if(i%n==0)
printf("\n");
printf("%c ",B[i]);
}

hipMalloc(&cudaA,size);;
hipMalloc(&cudaB,sizeof(char)*m*n);
hipMalloc(&cudaC,sizeof(char)*1024);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
hipMemcpy(cudaB,B,sizeof(char)*m*n,hipMemcpyHostToDevice);
dim3 dimGrid(1,1,1);
dim3 dimBlock(m,n,1);
calc<<<dimGrid,dimBlock>>>(cudaA,cudaB,cudaC,m,n);
hipMemcpy(C,cudaC,1024*sizeof(char),hipMemcpyDeviceToHost);
printf("\nMofified Matrix\n");
int k=strlen(C);
C[k]='\0';
for(int i=0;i<k;i++)
{
printf("%c",C[i]);
}
}
