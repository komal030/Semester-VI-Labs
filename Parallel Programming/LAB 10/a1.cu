#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void calc(int *A,int* B,int m,int n)
{
int row_id=threadIdx.x;
int col_id=threadIdx.y;
int sum=0;
for(int i=0;i<n;i++)
sum+=A[row_id*n+i];
for(int i=0;i<m;i++)
sum+=A[i*n+col_id];
B[row_id*n+col_id]=sum;
}


int main()
{
int m,n;
int *A;
printf("Enter number of rows:");
scanf("%d",&m);
printf("Enter number of columns:");
scanf("%d",&n);
int size=m*n*sizeof(int);
A=(int*)malloc(size);
printf("Enter the elements\n");
for(int i=0;i<m*n;i++)
{
scanf("%d",&A[i]);
}

printf("Original Matrix\n");
for(int i=0;i<m*n;i++)
{
if(i%n==0)
printf("\n");
printf("%d ",A[i]);
}
int *B;
int *cudaA;
int *cudaB;
hipMalloc(&cudaA,size);;
hipMalloc(&cudaB,size);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
dim3 dimGrid(1,1,1);
dim3 dimBlock(m,n,1);
calc<<<dimGrid,dimBlock>>>(cudaA,cudaB,m,n);
hipMemcpy(B,cudaB,size,hipMemcpyDeviceToHost);
printf("\nMofified Matrix\n");
for(int i=0;i<m*n;i++)
{
if(i%n==0)
printf("\n");
printf("%d ",B[i]);
}
}
