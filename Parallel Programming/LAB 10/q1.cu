#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void csr(int *data,int* col,int *row,int *x,int* res)
{
int id=threadIdx.x;
int start=row[id];
int end=row[id+1];
int sum=0;
for(int i=start;i<end;i++)
{
sum+=data[i]*X[col[i]];
}
res[id]=sum;
}

int main()
{
int data1[]={6,3,4,1,3,4,1,6};
int col1[]={1,2,3,0,1,3,2,3};
int row1[]={0,3,6,6,8};
int *data,*row,*col,*res,*x;
data=(int*)malloc(sizeof(int)*8);
col=(int*)malloc(sizeof(int)*8);
row=(int*)malloc(sizeof(int)*5);
res=(int*)malloc(sizeof(int)*4);
x=(int*)malloc(sizeof(int)*4);
for(int i=0;i<8;i++)
{
data[i]=data1[i];
col[i]=col1[i];
}
for(int i=0;i<5;i++)
{
row[i]=row1[i];
}
printf("Enter X values:\n");
for(int i=0;i<4;i++)
scanf("%d",&x[i]);
int *cudaData,*cudaRow,*cudaCol,*cudaRes,cudaX;
hipMalloc(&cudaData,sizeof(int)*8);
hipMalloc(&cudaRow,sizeof(int)*5);
hipMalloc(&cudaCol,sizeof(int)*8);
hipMalloc(&cudaRes,sizeof(int)*4);
hipMalloc(&cudaX,sizeof(int)*4);
hipMemcpy(cudaData,data,sizeof(int)*8,hipMemcpyHostToDevice);
hipMemcpy(cudaCol,col,sizeof(int)*8,hipMemcpyHostToDevice);
hipMemcpy(cudaRow,row,sizeof(int)*5,hipMemcpyHostToDevice);
hipMemcpy(cudaX,x,sizeof(int)*4,hipMemcpyHostToDevice);
csr<<<1,4>>>(cudaData,cudaCol,cudaRow,cudaX,cudaRes);
hipMemcpy(res,cudaRes,sizeof(int)*4,hipMemcpyDeviceToHost);
for(int i=0;i<4;i++)
{
printf("%d ",res[i]);
}
