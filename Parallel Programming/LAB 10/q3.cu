#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__device__ int findbin(int n)
{
int bin=0;
int pow=1;
while(n!=0)
{
int dig=n%2;
bin=bin+dig*pow;
n/=2;
pow*=10;
}
return bin;
}

__device__ int findones(int n)
{
int sum=0;
int pow=1;
while(n!=0)
{
int dig=n%10;
if(dig==1)
sum=sum+0*pow;
else
sum=sum+1*pow;
pow*=10;
n/=10;
}
return sum;

}


__global__ void border(int* A,int m,int n)
{
int row_id=threadIdx.x;
for(int i=0;i<n;i++)
{
if(i!=0 && row_id!=0 && row_id!=m-1 && i!= n-1)
{
int bin=findbin(A[row_id*n+i]);
int ones=findones(bin);
A[row_id*n+i]=ones;
}
}
}



int main()
{
int m,n;
int *A;
printf("Enter number of rows:");
scanf("%d",&m);
printf("Enter number of columns:");
scanf("%d",&n);
int size=m*n*sizeof(int);
A=(int*)malloc(size);
printf("Enter the elements\n");
for(int i=0;i<m*n;i++)
{
scanf("%d",&A[i]);
}

printf("Original Matrix\n");
for(int i=0;i<m*n;i++)
{
if(i%n==0)
printf("\n");
printf("%d ",A[i]);
}
int *cudaA;
hipMalloc(&cudaA,size);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
dim3 dimGrid(1,1,1);
dim3 dimBlock(m,1,1);
border<<<dimGrid,dimBlock>>>(cudaA,m,n);
hipMemcpy(A,cudaA,size,hipMemcpyDeviceToHost);
printf("\nMofified Matrix\n");
for(int i=0;i<m*n;i++)
{
if(i%n==0)
printf("\n");
printf("%d ",A[i]);
}
}

