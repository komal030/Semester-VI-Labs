#include "hip/hip_runtime.h"
%%writefile q1.cu
#include <stdio.h>
#include <stdlib.h>

__global__ void oddEvenSort(int *arr, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
        for (int i = 0; i < n; i++) {
            if (i % 2 == 0) { // Even phase
                if (idx % 2 == 0 && idx < n - 1) {
                    if (arr[idx] > arr[idx + 1]) {
                        int temp = arr[idx];
                        arr[idx] = arr[idx + 1];
                        arr[idx + 1] = temp;
                    }
                }
            } else { // Odd phase
                if (idx % 2 == 1 && idx < n - 1) {
                    if (arr[idx] > arr[idx + 1]) {
                        int temp = arr[idx];
                        arr[idx] = arr[idx + 1];
                        arr[idx + 1] = temp;
                    }
                }
            }
            __syncthreads(); // Synchronize threads after each phase
        }
}

int main() {
    int n = 8;
    int *A;
    int *cudaA;
    int size = n * sizeof(int);
    A = (int*)malloc(size);
    hipMalloc(&cudaA, size);
    for (int i = n; i >= 0; i--)
        A[n - i] = i;
    printf("Initial Array:\n");
    for (int i = 0; i < n; i++)
        printf("%d ", A[i]);
    hipMemcpy(cudaA, A, size, hipMemcpyHostToDevice);
    oddEvenSort<<<1, n>>>(cudaA, n);
    hipMemcpy(A, cudaA, size, hipMemcpyDeviceToHost);
    printf("\nSorted Array:\n");
    for (int i = 0; i < n; i++)
        printf("%d ", A[i]);
    printf("\n");
    free(A);
    hipFree(cudaA);
    return 0;
}

