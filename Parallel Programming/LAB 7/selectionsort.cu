#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

_global_ void sort(int *A,int *B,int n)
{
int id=threadIdx.x;
int nums=A[id];
int pos=0;
for(int i=0;i<n;i++)
{
if(A[i]<nums||(A[i]==nums && i<id))
{
pos++;
}
}
B[pos]=nums;
}

int main()
{
int *A,*B,n;
printf("Enter number of elements:\n");
scanf("%d",&n);
int size=n*sizeof(int);
A=(int*)malloc(size);
B=(int*)malloc(size);
printf("Enter the elements:\n");
for(int i=0;i<n;i++)
scanf("%d",&A[i]);
int* cudaA,*cudaB;
hipMalloc(&cudaA,size);
hipMalloc(&cudaB,size);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
sort<<<1,n>>>(cudaA,cudaB,n);
hipMemcpy(B,cudaB,size,hipMemcpyDeviceToHost);
printf("Sorted Array:\n");
for(int i=0;i<n;i++)
printf("%d ",B[i]);
}
