#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void sort(int *A,int *B,int n)
{
int id=threadIdx.x+blockIdx.x*blockDim.x;
int nums=A[id];
int pos=0;
for(int i=0;i<n;i++)
{
if(A[i]<nums||(A[i]==nums && i<id))
{
pos++;
}
}
B[pos]=nums;
}

int main()
{
int *A,*B,n,m;
m=3;
n=4;
int size=n*m*sizeof(int);
A=(int*)malloc(size);
B=(int*)malloc(size);
for(int i=m*n;i>=0;i--)
A[m*n-i]=i;
printf("Given Array:\n");
for(int i=0;i<n*m;i++)
printf("%d ",A[i]);
int *cudaA,*cudaB;
hipMalloc(&cudaA,size);
hipMalloc(&cudaB,size);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
sort<<<m,n>>>(cudaA,cudaB,n);
hipMemcpy(B,cudaB,size,hipMemcpyDeviceToHost);
printf("Sorted Array:\n");
for(int i=0;i<n*m;i++)
printf("%d ",B[i]);
}
