#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void add(int* A,int* B,int* C)
{
int id=blockIdx.x*blockDim.x +threadIdx.x;
C[id]=A[id]+B[id];
}

int main()
{
printf("Enter the number of elements:");
int n;
scanf("%d",&n);
int *A,*B,*C;
int size=n*sizeof(int);
A=(int*)malloc(size);
B=(int*)malloc(size);
C=(int*)malloc(size);
int* cudaA,*cudaB,*cudaC;
hipMalloc(&cudaA,size);
hipMalloc(&cudaB,size);
hipMalloc(&cudaC,size);
printf("Enter the elements in Vector A:\n");
for(int i=0;i<n;i++)
scanf("%d",&A[i]);
printf("Enter the elements in Vector B:\n");
for(int i=0;i<n;i++)
scanf("%d",&B[i]);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
hipMemcpy(cudaB,B,size,hipMemcpyHostToDevice);
add<<<n,1>>>(cudaA,cudaB,cudaC);
hipMemcpy(C,cudaC,size,hipMemcpyDeviceToHost);
printf("Resultant Matrix\n");
for(int i=0;i<n;i++)
printf("%d ",C[i]);
hipFree(cudaA);
hipFree(cudaB);
hipFree(cudaC);
free(A);
free(B);
free(C);
}
