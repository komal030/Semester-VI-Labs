#include <stdio.h>
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <math.h>
__global__ void sinecomp(double *A,double *B)
{
int id=threadIdx.x +blockIdx.x*blockDim.x;
B[id]=sinf(A[id]);
}

int main()
{
printf("Enter the number of elements in A:");
int n;
scanf("%d",&n);
printf("Enter the elements of A:\n");
double *A;
double *B;
A=(double*)malloc(n*sizeof(double));
B=(double*)malloc(n*sizeof(double));
for(int i=0;i<n;i++)
scanf("%lf",&A[i]);
double *cudaA;
double *cudaB;
hipMalloc(&cudaA,n*sizeof(double));
hipMalloc(&cudaB,n*sizeof(double));
hipMemcpy(cudaA,A,n*sizeof(double),hipMemcpyHostToDevice);
sinecomp<<<1,n>>>(cudaA,cudaB);
hipMemcpy(B,cudaB,n*sizeof(double),hipMemcpyDeviceToHost);
printf("Sine of Array A elements:\n");
for(int i=0;i<n;i++)
printf("%lf ",B[i]);
}
