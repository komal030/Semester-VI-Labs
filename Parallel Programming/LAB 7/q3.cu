#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void convolution1D(int* M,int* N,int* R,int m,int n)
{
int id=threadIdx.x;
int start=id-(m/2);
int result=0;
for(int i=0;i<m;i++)
{
int index=start+i;
if(index>=0&&index<n)
result+=N[index]*M[i];
}
R[id]=result;
}



int main()
{
printf("Enter the size of array N:");
int m,n;
scanf("%d",&n);
printf("Enter the size of Mask array:");
scanf("%d",&m);
int *M,*N,*R;
M=(int*)malloc(m*sizeof(int));
N=(int*)malloc(n*sizeof(int));
R=(int*)malloc(n*sizeof(int));
printf("Enter the values of array N");
for(int i=0;i<n;i++)
scanf("%d",&N[i]);
printf("Enter the values of Mask array");
for(int i=0;i<m;i++)
scanf("%d",&M[i]);
int *cudaM,*cudaN,*cudaR;
hipMalloc(&cudaM,m*sizeof(int));
hipMalloc(&cudaN,n*sizeof(int));
hipMalloc(&cudaR,n*sizeof(int));
hipMemcpy(cudaN,N,n*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(cudaM,M,m*sizeof(int),hipMemcpyHostToDevice);
convolution1D<<<1,n>>>(cudaM,cudaN,cudaR,m,n);
hipMemcpy(R,cudaR,n*sizeof(int),hipMemcpyDeviceToHost);
printf("Resultant Array:\n");
for(int i=0;i<n;i++)
printf("%d ",R[i]);
}
