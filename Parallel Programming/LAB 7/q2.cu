#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void add(int *A,int *B,int *C,int N)
{
int id=threadIdx.x+blockIdx.x*blockDim.x;
if(id<N)
{
C[id]=A[id]+B[id];
}
}

int main()
{
printf("Enter the number of elements");
int n;
scanf("%d",&n);
int *A,*B,*C;
int size=n*sizeof(int);
A=(int*)malloc(size);
B=(int*)malloc(size);
C=(int*)malloc(size);
int *cudaA,*cudaB,*cudaC;
hipMalloc(&cudaA,size);
hipMalloc(&cudaB,size);
hipMalloc(&cudaC,size);
printf("Enter the elements of Vector A:\n");
for(int i=0;i<n;i++)
scanf("%d",&A[i]);
printf("Enter the elements of Vector B:\n");
for(int i=0;i<n;i++)
scanf("%d",&B[i]);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
hipMemcpy(cudaB,B,size,hipMemcpyHostToDevice);
add<<<ceil(double)(n/2),2>>>(cudaA,cudaB,cudaC,n);
hipMemcpy(C,cudaC,size,hipMemcpyDeviceToHost);
printf("Resultant matrix:\n");
for(int i=0;i<n;i++)
printf("%d ",C[i]);
hipFree(cudaA);
hipFree(cudaB);
hipFree(cudaC);
free(A);
free(B);
free(C);
return 0;
}
