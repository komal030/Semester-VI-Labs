#include "hip/hip_runtime.h"
%%writefile q1.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void linear(int* X,int *C,int* Y,int m)
{
int id=threadIdx.x+blockIdx.x*blockDim.x;
Y[id]=m*X[id]+C[id];
}


int main()
{
int m=10;
int *X,*C,*Y;
int n=3;
int size=n*sizeof(int);
X=(int*)malloc(size);
C=(int*)malloc(size);
Y=(int*)malloc(size);
for(int i=0;i<n;i++)
{
X[i]=i+10;
C[i]=i+20;
}
int *cudaX,*cudaC,*cudaY;
hipMalloc(&cudaX,size);
hipMalloc(&cudaC,size);
hipMalloc(&cudaY,size);
hipMemcpy(cudaC,C,size,hipMemcpyHostToDevice);
hipMemcpy(cudaX,X,size,hipMemcpyHostToDevice);
linear<<<1,n>>>(cudaX,cudaC,cudaY,m);
hipMemcpy(Y,cudaY,size,hipMemcpyDeviceToHost);
for(int i=0;i<n;i++)
printf("%d=10*%d+%d\n",Y[i],X[i],C[i]);
}
 
 
