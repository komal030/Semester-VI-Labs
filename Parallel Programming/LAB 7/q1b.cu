#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void add(int* A,int* B,int* C)
{
int id=threadIdx.x;
C[id]=A[id]+B[id];
}

int main()
{
int *A,*B,*C;
printf("Enter number of elements in vector: ");
int n;
scanf("%d",&n);
int size=n*sizeof(int);
A=(int*)malloc(size);
B=(int*)malloc(size);
C=(int*)malloc(size);
printf("Enter elements of Vector A:\n");
for(int i=0;i<n;i++)
scanf("%d",&A[i]);
printf("Enter elements of Vector B:\n");
for(int i=0;i<n;i++)
scanf("%d",&B[i]);
int *cudaA,*cudaB,*cudaC;
hipMalloc(&cudaA,size);
hipMalloc(&cudaB,size);
hipMalloc(&cudaC,size);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
hipMemcpy(cudaB,B,size,hipMemcpyHostToDevice);
add<<<1,n>>>(cudaA,cudaB,cudaC);
hipMemcpy(C,cudaC,size,hipMemcpyDeviceToHost);
printf("Resultant Vector:\n");
for(int i=0;i<n;i++)
printf("%d ",C[i]);
hipFree(cudaA);
hipFree(cudaB);
hipFree(cudaC);
free(A);
free(B);
free(C);
return 0;
}
