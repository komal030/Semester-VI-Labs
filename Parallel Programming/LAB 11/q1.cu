#include "hip/hip_runtime.h"
%%writefile q1.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

__constant__ int cudaN[100];
__constant__ int cudaM[100];
__constant__ int cudan;
__constant__ int cudam;

__global__ void convolution1d(int *Res)
{
int id=threadIdx.x;
int start=id-(cudam/2);
int sum=0;
for(int i=0;i<cudam;i++)
{
int ind=start+i;
if(ind>=0 && ind<cudan)
sum+=cudaN[ind]*cudaM[i];
}
Res[id]=sum;
}

int main()
{
int *N,*M,n,m,*Res,*cudaRes;
printf("Enter N:");
scanf("%d",&n);
printf("Enter M:");
scanf("%d",&m);
Res=(int*)malloc(sizeof(int)*n);
N=(int*)malloc(sizeof(int)*n);
M=(int*)malloc(sizeof(int)*m);
printf("Enter elements of N:\n");
for(int i=0;i<n;i++)
scanf("%d",&N[i]);
printf("Enter elements of M:\n");
for(int i=0;i<m;i++)
scanf("%d",&M[i]);
hipMalloc(&cudaRes,sizeof(int)*n);
hipMemcpyToSymbol(HIP_SYMBOL(cudaN),N,sizeof(int)*n);
hipMemcpyToSymbol(HIP_SYMBOL(cudaM),M,sizeof(int)*m);
hipMemcpyToSymbol(HIP_SYMBOL(cudan),&n,sizeof(int));
hipMemcpyToSymbol(HIP_SYMBOL(cudam),&m,sizeof(int));
convolution1d<<<1,n>>>(cudaRes);
hipMemcpy(Res,cudaRes,sizeof(int)*n,hipMemcpyDeviceToHost);
printf("\nGiven Array\n");
for(int i=0;i<n;i++)
printf("%d ",N[i]);

printf("\nMask Array\n");
for(int i=0;i<m;i++)
printf("%d ",M[i]);

printf("\nResultant Array\n");
for(int i=0;i<n;i++)
printf("%d ",Res[i]);
}

