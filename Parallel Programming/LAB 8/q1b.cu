#include "hip/hip_runtime.h"
%%writefile q1.cu
#include<stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
//columnwise-wise addition
__global__ void matadd(int *A,int *B,int *C)
{
int id=threadIdx.x+blockIdx.x*blockDim.x;
C[id]=A[id]+B[id];
}

int main()
{
int n=2,m=3;
int size=n*m*sizeof(int);
int *A,*B,*C;
A=(int*)malloc(size);
B=(int*)malloc(size);
C=(int*)malloc(size);
for(int i=0;i<m*n;i++)
{
A[i]=i+1;
B[i]=i+1;
}
int *cudaA,*cudaB,*cudaC;
hipMalloc(&cudaA,size);
hipMalloc(&cudaB,size);
hipMalloc(&cudaC,size);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
hipMemcpy(cudaB,B,size,hipMemcpyHostToDevice);
dim3 dimGrid(n,1,1);
dim3 dimBlock(m,1,1);
matadd<<<dimGrid,dimBlock>>>(cudaA,cudaB,cudaC);
hipMemcpy(C,cudaC,size,hipMemcpyDeviceToHost);
printf("Resultant:\n");
int c=0;
for(int i=0;i<m;i++)
{
for(int j=0;j<n;j++)
printf("%d ",C[i*c+j]);
printf("\n");
c++;
}
}

