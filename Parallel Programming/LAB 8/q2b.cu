#include "hip/hip_runtime.h"
%%writefile q1.cu
#include<stdlib.h>
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <>
//column-wise multiplication 
//m*n
//p*q
//m*q
__global__ void mulcol(int *A,int *B,int *C,int m,int n,int q)
{
int col_id=threadIdx.x;
int sum=0;
for(int row_id=0;row_id<m;row_id++)
{
sum=0;
for(int k=0;k<n;k++)
{
sum+=A[row_id*n+k]*B[k*q+col_id];
}
C[row_id*q+col_id]=sum;
}
}

int main()
{
int m=2,n=3,p=3,q=3;

int *A,*B,*C;
A=(int*)malloc(sizeof(int)*m*n);
B=(int*)malloc(sizeof(int)*p*q);
C=(int*)malloc(sizeof(int)*m*q);
for(int i=0;i<m*n;i++)
{
A[i]=i+1;
}
for(int i=0;i<p*q;i++)
{
B[i]=i+1;
}
int *cudaA,*cudaB,*cudaC;
hipMalloc(&cudaA,sizeof(int)*m*n);
hipMalloc(&cudaB,sizeof(int)*p*q);
hipMalloc(&cudaC,sizeof(int)*m*q);
hipMemcpy(cudaA,A,sizeof(int)*m*n,hipMemcpyHostToDevice);
hipMemcpy(cudaB,B,sizeof(int)*p*q,hipMemcpyHostToDevice);
dim3 dimGrid(1,1,1);
dim3 dimBlock(n,1,1);
mulcol<<<dimGrid,dimBlock>>>(cudaA,cudaB,cudaC,m,n,q);
hipMemcpy(C,cudaC,sizeof(int)*m*q,hipMemcpyDeviceToHost);
printf("Resultant:\n");
for(int i=0;i<m*q;i++)
{
  if(i%q==0)
  printf("\n");
printf("%d ",C[i]);

}
}



