#include "hip/hip_runtime.h"
%%writefile q1.cu
#include<stdlib.h>
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <>
//row-wise multiplication 
//m*n
//p*q
//m*q
__global__ void mulrow(int *A,int *B,int *C,int m,int n,int q)
{
int row_id=threadIdx.x;
for(int col_id=0;col_id<q;col_id++)
{
int sum=0;
for(int k=0;k<n;k++)
{
sum+=A[row_id*n+k]*B[k*q+col_id];
}
C[row_id*q+col_id]=sum;
}
}


int main()
{
int m=2,n=3,p=3,q=3;

int *A,*B,*C;
A=(int*)malloc(sizeof(int)*m*n);
B=(int*)malloc(sizeof(int)*p*q);
C=(int*)malloc(sizeof(int)*m*q);
for(int i=0;i<m*n;i++)
{
A[i]=i+1;
}
for(int i=0;i<p*q;i++)
{
B[i]=i+1;
}
int *cudaA,*cudaB,*cudaC;
hipMalloc(&cudaA,sizeof(int)*m*n);
hipMalloc(&cudaB,sizeof(int)*p*q);
hipMalloc(&cudaC,sizeof(int)*m*q);
hipMemcpy(cudaA,A,sizeof(int)*m*n,hipMemcpyHostToDevice);
hipMemcpy(cudaB,B,sizeof(int)*p*q,hipMemcpyHostToDevice);
dim3 dimGrid(1,1,1);
dim3 dimBlock(m,1,1);
mulrow<<<dimGrid,dimBlock>>>(cudaA,cudaB,cudaC,m,n,q);
hipMemcpy(C,cudaC,sizeof(int)*m*q,hipMemcpyDeviceToHost);
printf("Resultant:\n");
int c=0;
for(int i=0;i<m*q;i++)
{
  if(i%q==0)
  printf("\n");
printf("%d ",C[i]);

}
}



