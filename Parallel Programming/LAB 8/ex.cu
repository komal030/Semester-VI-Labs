#include "hip/hip_runtime.h"
%%writefile q1.cu
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void countalp(char *A,char target,int* count)
{
int id=threadIdx.x;
if(A[id]==target)
atomicAdd(count,1);
}

int main()
{
char *A;
A=(char*)malloc(1024*sizeof(char));
int count=0;
for(int i=0;i<10;i++)
A[i]=(char)i+95;
char *cudaA;
hipMalloc(&cudaA,strlen(A)*sizeof(char));
int *d_count;
hipMalloc(&d_count,sizeof(int));
char target='a';

hipMemcpy(d_count,&count,sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(cudaA,A,strlen(A)*sizeof(char),hipMemcpyHostToDevice);
countalp<<<1,strlen(A)>>>(cudaA,target,d_count);
hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);
printf("Count is %d",count);
}
