#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void transpose(int* A,int* T)
{
int row_id=blockIdx.x;
int col_id=threadIdx.x;
int m=gridDim.x;
int n=blockDim.x;
T[row_id*m+col_id]=A[row_id*n+col_id]
}

int main()
{
int *A,*T;
int m=2,n=3;
int size=m*n*sizeof(int);
A=(int*)malloc(size);
T=(int*)malloc(size);
for(int i=0;i<n*m;i++)
A[i]=i+1;
int *cudaA,*cudaT;
hipMalloc(cudaA,size);
hipMalloc(cudaT,size);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice):
transpose<<<m,n>>>(cudaA,cudaT);
hipMemcpy(T,cudaT,size,hipMemcpyDeviceToHost);
printf("Transpose\n");
for(int i=0;i<m*n;i++)
{
if(i%n==0)
printf("\n");
printf("%d ",A[i]);
}
printf("Transpose\n");
for(int i=0;i<m*n;i++)
{
if(i%m==0)
printf("\n");
printf("%d ",T[i]);
}
}
