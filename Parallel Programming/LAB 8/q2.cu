#include "hip/hip_runtime.h"
%%writefile q1.cu
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void calc(char *A,char *B,int n)
{
int id = threadIdx.x;
	int size = n - id;
	int ind = (id * (size + 1 + n)) / 2;
	for(int i = 0; i < size; i++){
		B[ind + i] = A[i];
	}
}

int main()
{
char  *A=(char*)malloc(sizeof(char)*1024);
for(int i=0;i<4;i++)
A[i]=char(65+i);
int n=strlen(A);
int m= (n*(n+1))/2;
char *cudaB,*cudaA;
printf("m=%d\n",m);
printf("n=%d\n",n);
printf("a=%s\n",A);
char *B=(char*)malloc(sizeof(char)*1024);
hipMalloc(&cudaA,sizeof(char)*n);
hipMalloc(&cudaB,sizeof(char)*m);
hipMemcpy(cudaA,A,sizeof(char)*n,hipMemcpyHostToDevice);
calc<<<1,n>>>(cudaA,cudaB,n);
hipMemcpy(B,cudaB,sizeof(char)*m,hipMemcpyDeviceToHost);
printf("%s",B);
}
