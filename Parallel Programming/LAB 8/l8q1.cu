#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>


__global__ void countocc(char* A,char* B,int* pos,int* count,int wcount)
{
int id=threadIdx.x;
int start=pos[id*2];
int end=pos[(id*2)+1];
int j=0,flag=1;
for(int i=start;i<=end;i++,j++)
{
if(j<wcount && A[i]!=B[j])
{
return;
}
}
if(j==wcount)
atomicAdd(count,1);
}





int main()
{
char *A=(char*)malloc(sizeof(char)*1024);
char *cudaA,*cudaB;
char *B=(char*)malloc(sizeof(char)*1024);
printf("Enter the string\n");
fgets(A,1024,stdin);
printf("Enter the word\n");
scanf("%s",B);
int n=strlen(A);
int m=strlen(B);
A[n]='\0';
B[m]='\0';
int* pos=(int*)malloc(sizeof(int)*100);
int k=1;
pos[0]=0;
for(int  i=0;i<n;i++)
{
if(A[i]==' ')
{
pos[k++]=i-1;
pos[k++]=i+1;
}
}
pos[k++]=n-1;
pos[k]='\0';
for(int i=0;i<k;i++)
printf("%d\n",pos[i]);
int count=0,*cudaCount;
int *hipPos;
hipMalloc(&cudaA,sizeof(char)*n);
hipMalloc(&cudaB,sizeof(char)*m);
hipMalloc(&cudaCount,sizeof(int));
hipMalloc(&hipPos,sizeof(int)*k);
hipMemcpy(cudaA,A,sizeof(char)*n,hipMemcpyHostToDevice);
hipMemcpy(cudaB,B,sizeof(char)*m,hipMemcpyHostToDevice);
//cudaMemcpy(cudaB,B,sizeof(char)*m,cudaMemcpyHostToDevice);
hipMemcpy(hipPos,pos,sizeof(int)*k,hipMemcpyHostToDevice);
//printf("%d",k/2);
countocc<<<1,k/2>>>(cudaA,cudaB,hipPos,cudaCount,m);
hipMemcpy(&count,cudaCount,sizeof(int),hipMemcpyDeviceToHost);
printf("Count of %s is %d\n",B,count);
}
