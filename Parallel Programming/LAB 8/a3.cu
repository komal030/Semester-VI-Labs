#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <string.h>

__global__ void copystr(char *A,char* B,int n )
{
int id=threadIdx.x;//1
int index=((id+1)*id)/2;
for(int i=0;i<id+1;i++)
B[index+i]=A[id];
}

int main()
{
char *A;
A=(char*)malloc(sizeof(char)*1024);
printf("Enter String:\n");
fgets(A,1024,stdin);
char *B;
B=(char*)malloc(sizeof(char)*1024);
char *cudaA,*cudaB;
int n=strlen(A);
A[n]='\0';
hipMalloc(&cudaA,n*sizeof(char));
int m=((n+1)*n)/2;
hipMalloc(&cudaB,m*sizeof(char));
hipMemcpy(cudaA,A,sizeof(char)*n,hipMemcpyHostToDevice);
copystr<<<1,n>>>(cudaA,cudaB,n);
hipMemcpy(B,cudaB,sizeof(char)*m,hipMemcpyDeviceToHost);
B[m]='\0';
printf("A=%s\n",A);
printf("B=%s\n",B);
}
