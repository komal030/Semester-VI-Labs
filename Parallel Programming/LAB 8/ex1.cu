#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void getFrequency(char* str, int* count, char target){
	int i = threadIdx.x;
	if(str[i] == target){
		atomicAdd(count, 1);
	}
	return;
}

int main(){
	char* str = (char*)malloc(sizeof(char) * 1024);
	char* cudaStr;
	int* count = 0;
	int* cudaCount = 0;
	int* result = (int*)malloc(sizeof(int));
	char target;
	printf("\nEnter a string: ");
	fgets(str, 1024, stdin);
	printf("\nEnter target character: ");
	scanf("%c", &target);
	hipMalloc(&cudaStr, strlen(str) * sizeof(char));
	hipMalloc(&cudaCount, sizeof(int));
	hipMemcpy(cudaStr, str, strlen(str) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(cudaCount, count, sizeof(int), hipMemcpyHostToDevice);
	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(strlen(str), 1, 1);
	getFrequency <<<dimGrid, dimBlock>>> (cudaStr, cudaCount, target);
	hipMemcpy(result, cudaCount, sizeof(int), hipMemcpyDeviceToHost);
	printf("\nTotal Occurences of %c is %d", target, *result);
	return 0;
}
