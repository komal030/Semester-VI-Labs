#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void countocc(char* A,char* B,int pos,int* count,int wcount)
{
int id=threadIdx.x;
int start=pos[id*2];
int end=pos[id*2+1];
int j=0;
for(int i=start;i<=end;i++)
{
if(j<wcount && A[i]==B[j])
{
j++;
}
else
return;
}
atomicAdd(count,1);
}

int main()
{
char *A=(char*)malloc(sizeof(char)*1024);
char *cudaA,*cudaB;
char *B=(char*)malloc(sizeof(char)*1024);
printf("Enter the string\n");
fgets(A,1024,stdin);
printf("Enter the word\n");
scanf("%s",B);
int n=strlen(A);
int m=strlen(B);
int pos=(int*)malloc(sizeof(int)*100);
int k=0,start=0,wordcount=0;
for(int  i=0;i<=n;i++)
{
if(A[i]==' '||A[i]=='\0')
{
wordcount++;
pos[k++]=start;
pos[k++]=i-1;
start=i+1;
}
}
int count=0,*cudaCount;
hipMalloc(&cudaA,sizeof(char)*n);
hipMalloc(&cudaB,sizeof(char)*m);
hipMalloc(&cudaCount,sizeof(int));
hipMemcpy(cudaA,A,sizeof(char)*n,hipMemcpyHostToDevice);
hipMemcpy(cudaB,B,sizeof(char)*m,hipMemcpyHostToDevice);
dim3 dimGrid(1,1,1);
dim3 dimBlock(wordcount,1,1);
countocc<dimGrid,dimBlock>>>(cudaA,cudaB,pos,cudaCount,m);
hipMemcpy(&count,cudaCount,sizeof(int),hipMemcpyDeviceToHost);
printf("Count of %s is %d\n",B,count);
}
