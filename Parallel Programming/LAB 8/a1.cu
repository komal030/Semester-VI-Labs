#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void calc(int *A,int *B,int m,int n)
{
int row_id=threadIdx.x;
int col_id=threadIdx.y;
int sum=0;
for(int i=0;i<m;i++)
sum+=A[n*i+col_id];
for(int i=0;i<n;i++)
sum+=A[row_id*n+i];
B[row_id*n+col_id]=sum;
}

int main()
{
int m,n;
printf("Enter m:");
scanf("%d",&m);
printf("Enter n:");
scanf("%d",&n);
int size=sizeof(int)*m*n;
int *A,*B;
int *cudaA,*cudaB;
A=(int*)malloc(size);
B=(int*)malloc(size);
printf("Enter matrix elements");
for(int i=0;i<m*n;i++)
scanf("%d",&A[i]);
hipMalloc(&cudaA,size);
hipMalloc(&cudaB,size);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
dim3 dimGrid(1,1,1);
dim3 dimBlock(m,n,1);
calc<<<dimGrid,dimBlock>>>(cudaA,cudaB,m,n);
hipMemcpy(B,cudaB,size,hipMemcpyDeviceToHost);
for(int i=0;i<m*n;i++)
{
if(i%n==0)
printf("\n");
printf("%d ",B[i]);
}
}
