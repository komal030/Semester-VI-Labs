#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void reverse(char *A,int* pos)
{
int id=threadIdx.x;
int start=pos[2*id];
int end=pos[2*id+1];

while(start<=end)
{
char temp=A[start];
A[start]=A[end];
A[end]=temp;
start++;
end--;
}
}

int main()
{
char *A;
int *pos;
A=(char*)malloc(sizeof(char)*1024);
printf("Enter sentence\n");
fgets(A,1024,stdin);
pos=(int*)malloc(sizeof(int)*100);
pos[0]=0;
int n=strlen(A);
int size=n*sizeof(char);
int k=1;
for(int i=0;i<n;i++)
{
if(A[i]==' ')
{
pos[k++]=i-1;
pos[k++]=i+1;
}
}
pos[k++]=n-1;
pos[k]='\0';
int *hipPos;
char *cudaA;
hipMalloc(&cudaA,size);
hipMalloc(&hipPos,sizeof(int)*k);
printf("A= %s\n",A);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
hipMemcpy(hipPos,pos,sizeof(int)*k,hipMemcpyHostToDevice);
reverse<<<1,k/2>>>(cudaA,hipPos);
hipMemcpy(A,cudaA,size,hipMemcpyDeviceToHost);

printf("R= %s\n",A);
}
