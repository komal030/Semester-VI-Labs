#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>


__global__ void repeatchar(char *A,char *B,int n)
{
int id=threadIdx.x;
int a_size=blockDim.x;
for(int i=0;i<n;i++)
B[i*a_size+id]=A[id];
}

int main()
{
char *A,*B,*cudaA,*cudaB;
A=(char*)malloc(sizeof(char)*1024);
B=(char*)malloc(sizeof(char)*1024);
for(int i=0;i<4;i++)
A[i]=char(i+65);
int n=5;
int m=n*strlen(A);
hipMalloc(&cudaA,strlen(A)*sizeof(char));
hipMalloc(&cudaB,m*sizeof(char));
hipMemcpy(cudaA,A,strlen(A)*sizeof(char),hipMemcpyHostToDevice);
repeatchar<<<1,strlen(A)>>>(cudaA,cudaB,n);
hipMemcpy(B,cudaB,m*sizeof(char),hipMemcpyDeviceToHost);
printf("A=%s\n",A);
printf("B=%s\n",B);
}
