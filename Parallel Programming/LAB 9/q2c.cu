#include "hip/hip_runtime.h"
%%writefile q1.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void mulele(int *A,int *B,int *C,int m,int n,int q)
{
int row_id=threadIdx.x;
int col_id=threadIdx.y;
int sum=0;
for(int k=0;k<n;k++)
{
sum+=A[row_id*n+k]*B[k*q+col_id];
}
C[row_id*q+col_id]=sum;
}

int main()
{
int *A,*B,*C;
int m=2,n=3,p=3,q=3;
A=(int*)malloc(m*n*sizeof(int));
B=(int*)malloc(p*q*sizeof(int));
C=(int*)malloc(m*q*sizeof(int));
for(int i=0;i<m*n;i++)
{
A[i]=i+1;
}
for(int i=0;i<p*q;i++)
{
B[i]=i+1;
}
int *cudaA,*cudaB,*cudaC;
hipMalloc(&cudaA,m*n*sizeof(int));
hipMalloc(&cudaB,p*q*sizeof(int));
hipMalloc(&cudaC,m*q*sizeof(int));
hipMemcpy(cudaA,A,m*n*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(cudaB,B,p*q*sizeof(int),hipMemcpyHostToDevice);
dim3 dimGrid(1,1,1);
dim3 dimBlock(m,q,1);
mulele<<<dimGrid,dimBlock>>>(cudaA,cudaB,cudaC,m,n,q);
hipMemcpy(C,cudaC,m*q*sizeof(int),hipMemcpyDeviceToHost);
printf("Resultant:\n");
int c=0;
for(int i=0;i<m;i++)
{
for(int j=0;j<n;j++)
printf("%d ",C[n*c+j]);
printf("\n");
c++;
}
}



