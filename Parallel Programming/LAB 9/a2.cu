#include "hip/hip_runtime.h"
%%writefile q1.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

__device__ int factorial(int n)
{
int f=1;
for(int i=1;i<=n;i++)
f=f*i;
return f;
}

__device__ int sumofdig(int n)
{
int sum=0;
while(n!=0)
{
sum+=n%10;
n/=10;
}
return sum;
}



__global__ void compute(int *A,int *B,int n)
{
int row_id=threadIdx.x;
for(int i=0;i<n;i++)
{
if(i==row_id)
B[row_id*n+i]=0;
else if(row_id>i)
{
int f=factorial(A[row_id*n+i]);
B[row_id*n+i]=f;
}
else
{
int sum=sumofdig(A[row_id*n+i]);
B[row_id*n+i]=sum;
}
}
}


int main()
{
int *A,*B;
int n=3;
int *cudaA,*cudaB;
int size=n*n*sizeof(int);
A=(int*)malloc(size);
B=(int*)malloc(size);
hipMalloc(&cudaA,size);
hipMalloc(&cudaB,size);
for(int i=0;i<n*n;i++)
{
A[i]=i+1;
}
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
compute<<<1,n>>>(cudaA,cudaB,n);
hipMemcpy(B,cudaB,size,hipMemcpyDeviceToHost);
printf("Given Matrix:\n");
for(int i=0;i<n*n;i++)
{
if(i%n==0)
printf("\n");
printf("%d ",A[i]);
}

printf("Resultant Matrix:\n");
for(int i=0;i<n*n;i++)
{
if(i%n==0)
printf("\n");
printf("%d ",B[i]);
}
hipFree(cudaA);
hipFree(cudaB);
free(A);
free(B);
}


