#include<stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//each element addition with single thread
__global__ void matadd(int *A,int *B,int *C)
{
int id=threadIdx.x;
C[id]=A[id]+B[id];
}

int main()
{
int n=2,m=3;
int size=n*m*sizeof(int);
int *A,*B,*C;
A=(int*)malloc(size);
B=(int*)malloc(size);
C=(int*)malloc(size);
for(int i=0;i<m*n;i++)
{
A[i]=i+1;
B[i]=i+1;
}
int *cudaA,*cudaB,*cudaC;
hipMalloc(&cudaA,size);
hipMalloc(&cudaB,size);
hipMalloc(&cudaC,size);
hipMemcpy(cudaA,A,size,hipMemcpyHostToDevice);
hipMemcpy(cudaB,B,size,hipMemcpyHostToDevice);
matadd<<<1,m*n>>>(cudaA,cudaB,cudaC);
hipMemcpy(C,cudaC,size,hipMemcpyDeviceToHost);
printf("Resultant:\n");
int c=0;
for(int i=0;i<m;i++)
{
for(int j=0;j<n;j++)
printf("%d ",C[n*c+j]);
printf("\n");
c++;
}
}

